#include "cuda_malloc.h"
#include "hip/hip_runtime_api.h"    /* --> checkCudaErrors */


void cuda_malloc(void** devPtr, size_t size) {
  checkCudaErrors(hipMalloc(devPtr, size));
}

void cuda_calloc(void** devPtr, size_t size) {
  hipError_t cudaCalloc_er = hipMalloc(devPtr, size);
  if (cudaCalloc_er == hipSuccess) {
    return hipMemset(*devPtr, 0, size);
  }
  else {
    return cudaCalloc_er;
  }
}

void cuda_free(void** devPtr) {
  hipError_t cuda_error = hipFree(*devPtr);
  *devPtr = NULL;
  return cuda_error;
}
