#include "cuda_malloc.h"
#include "hip/hip_runtime_api.h"    /* --> checkCudaErrors */


#define c_cudaMalloc hipMalloc

template<typename T>
inline hipError_t  c_cudaCalloc(T** devPtr, size_t size) {
  hipError_t cudaCalloc_er = hipMalloc(devPtr, size);
  if (cudaCalloc_er == hipSuccess) {
    return hipMemset(*devPtr, 0, size);
  }
  else {
    return cudaCalloc_er;
  }
}

template<typename T>
inline hipError_t c_cudaFree(T** devPtr) {
  hipError_t cuda_error = hipFree(*devPtr);
  *devPtr = NULL;
  return cuda_error;
}


void cuda_malloc(void** devPtr, size_t size) {
  checkCudaErrors(c_cudaMalloc(devPtr, size));
}

void cuda_calloc(void** devPtr, size_t size) {
  checkCudaErrors(c_cudaCalloc(devPtr, size));
}

void cuda_free(void** devPtr, size_t size) {
  c_cudaFree(devPtr);
}
