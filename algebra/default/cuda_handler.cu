#include "cuda_handler.h"
#include "hip/hip_runtime_api.h"


CUDA_Handle_t CUDA_handle;
CUDA_handle.cublasHandle   = NULL;
CUDA_handle.cusparseHandle = NULL;
CUDA_handle.d_index        = NULL;


void CUDA_init_libs(void) {

  int deviceCount = 0;

  hipGetDeviceCount(&deviceCount);
  if (!deviceCount) printf("No GPU detected.\n");

  checkCudaErrors(hipSetDevice(0));
  checkCudaErrors(hipsparseCreate(&CUDA_handle.cusparseHandle));
  checkCudaErrors(hipblasCreate(&CUDA_handle.cublasHandle));
  checkCudaErrors(hipMalloc(&CUDA_handle.d_index, sizeof(int)));
}


void CUDA_free_libs() {
  hipsparseDestroy(CUDA_handle.cusparseHandle);
  hipblasDestroy(CUDA_handle.cublasHandle);
  hipFree(CUDA_handle.d_index);
}
