#include "cuda_handler.h"
#include "hip/hip_runtime_api.h"


CUDA_Handle_t* CUDA_init_libs(void) {

  int deviceCount = 0;

  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0) {
    printf("No GPU detected.\n");
    return NULL;
  }

  CUDA_Handle_t *CUDA_handle = (CUDA_Handle_t*) malloc(sizeof(CUDA_Handle_t));
  if (!CUDA_handle) {
    printf("Memory allocation error.\n");
    return NULL;
  }

  checkCudaErrors(hipSetDevice(0));
  checkCudaErrors(hipsparseCreate(&CUDA_handle->cusparseHandle));
  checkCudaErrors(hipblasCreate(&CUDA_handle->cublasHandle));
  checkCudaErrors(hipMalloc(&CUDA_handle->d_index, sizeof(int)));

  return CUDA_handle;
}


void CUDA_free_libs(CUDA_Handle_t *CUDA_handle) {
  hipsparseDestroy(CUDA_handle->cusparseHandle);
  hipblasDestroy(CUDA_handle->cublasHandle);
  hipFree(CUDA_handle->d_index);
  free(CUDA_handle);
}

