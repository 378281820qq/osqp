#include "cuda_malloc.h"
#include "hip/hip_runtime_api.h"    /* --> checkCudaErrors */


#define c_cudaMalloc hipMalloc
#define c_cudaMallocHost hipHostMalloc


template<typename T>
inline hipError_t  c_cudaCalloc(T** devPtr, size_t size) {
  hipError_t cudaCalloc_er = hipMalloc(devPtr, size);
  if (cudaCalloc_er == hipSuccess) {
    return hipMemset(*devPtr, 0, size);
  }
  else {
    return cudaCalloc_er;
  }
}

template<typename T>
inline hipError_t c_cudaFree(T** devPtr) {
  hipError_t cuda_error = hipFree(*devPtr);
  *devPtr = NULL;
  return cuda_error;
}

template<typename T>
inline hipError_t c_cudaFreeHost(T** devPtr) {
  hipError_t cuda_error = hipHostFree(*devPtr);
  *devPtr = NULL;
  return cuda_error;
}


void cuda_malloc(void** devPtr, size_t size) {
  checkCudaErrors(c_cudaMalloc(devPtr, size));
}

void cuda_malloc_host(void** devPtr, size_t size) {
  checkCudaErrors(c_cudaMallocHost(devPtr, size));
}

void cuda_calloc(void** devPtr, size_t size) {
  checkCudaErrors(c_cudaCalloc(devPtr, size));
}

void cuda_free(void** devPtr) {
  checkCudaErrors(c_cudaFree(devPtr));
}

void cuda_free_host(void** devPtr) {
  checkCudaErrors(c_cudaFreeHost(devPtr));
}
