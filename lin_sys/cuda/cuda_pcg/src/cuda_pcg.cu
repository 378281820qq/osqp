#include "cuda_pcg.h"
#include "csr_type.h"
#include "cuda_handler.h"
#include "cuda_malloc.h"
#include "cuda_lin_alg.h"
#include "cuda_wrapper.h"
#include "hip/hip_runtime_api.h"    /* --> checkCudaErrors */

extern CUDA_Handle_t *CUDA_handle;


void cuda_pcg_update_precond(cudapcg_solver *s,
                             c_int           P_updated,
                             c_int           A_updated,
                             c_int           R_updated) {

  void    *buffer;
  c_float *tmp;
  c_int    n  = s->n;
  csr     *At = s->At;

  size_t Buffer_size_in_bytes = n * (sizeof(c_float) + sizeof(c_int));

  if (!P_updated && !A_updated && !R_updated) return;

  if (R_updated) cuda_vec_copy_h2d(s->d_rho, s->h_rho, 1);

  if (P_updated) {
    /* Update d_P_diag_val */
    checkCudaErrors(cusparseTgthr(CUDA_handle->cusparseHandle, n, s->P->val, s->d_P_diag_val, s->d_P_diag_ind, HIPSPARSE_INDEX_BASE_ZERO));
  }

  if (A_updated || R_updated) {
    /* Allocate memory */
    cuda_malloc((void **) &tmp, At->nnz * sizeof(c_float));
    cuda_malloc((void **) &buffer, Buffer_size_in_bytes);

    /* Update d_AtRA_diag_val */
    if (!s->d_rho_vec) {  /* R = rho*I  -->  A'*R*A = rho * A'*A */

      if (A_updated) {
        /* Update d_AtA_diag_val */
        cuda_vec_ew_prod(tmp, At->val, At->val, At->nnz);
        cuda_vec_segmented_sum(tmp, At->row_ind, s->d_AtA_diag_val, buffer, n, At->nnz);
      }

      /* d_AtRA_diag_val = rho * d_AtA_diag_val */
      cuda_vec_add_scaled(s->d_AtRA_diag_val, s->d_AtA_diag_val, NULL, *s->h_rho, 0.0, n);
    }
    else {    /* R = diag(d_rho_vec)  -->  A'*R*A = A' * diag(d_rho_vec) * A */
      cuda_mat_rmult_diag_new(At, tmp, s->d_rho_vec);   /* tmp = A' * R */
      cuda_vec_ew_prod(tmp, tmp, At->val, At->nnz);     /* tmp = tmp * A */
      cuda_vec_segmented_sum(tmp, At->row_ind, s->d_AtRA_diag_val, buffer, n, At->nnz);
    }

    /* Free memory */
    cuda_free((void **) &tmp);
    cuda_free((void **) &buffer);
  }

  /* d_diag_precond = sigma */
  cuda_vec_set_sc(s->d_diag_precond, *s->h_sigma, n);

  /* d_diag_precond += d_P_diag_val + d_AtRA_diag_val */
  cuda_vec_add_scaled3(s->d_diag_precond, s->d_diag_precond, s->d_P_diag_val, s->d_AtRA_diag_val, 1.0, 1.0, 1.0, n);

  /* d_diag_precond_inv = 1 / d_diag_precond */
  cuda_vec_reciprocal(s->d_diag_precond_inv, s->d_diag_precond, n);
}
